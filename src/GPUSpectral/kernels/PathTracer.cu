#include "hip/hip_runtime.h"
#include <optix.h>
#include <hip/hip_vector_types.h>
#include <vector_functions.hpp>
#include "PathTracer.cuh"
#include "VectorMath.cuh"
#include <math.h>

extern "C" {
    __constant__ Params params;
}

struct RadiancePRD {
    float3       emitted;
    float3       radiance;
    float3       weight;
    float3       origin;
    float3       direction;
    
    SamplerState sampler;
    int          wasDelta;
    int          countEmitted;
    int          done;
    int          pad;
};

static __forceinline__ __device__ void* unpackPointer( unsigned int i0, unsigned int i1 )
{
    const unsigned long long uptr = static_cast<unsigned long long>( i0 ) << 32 | i1;
    void*           ptr = reinterpret_cast<void*>( uptr );
    return ptr;
}

static __forceinline__ __device__ void  packPointer( void* ptr, unsigned int& i0, unsigned int& i1 )
{
    const unsigned long long uptr = reinterpret_cast<unsigned long long>( ptr );
    i0 = uptr >> 32;
    i1 = uptr & 0x00000000ffffffff;
}

static __forceinline__ __device__ RadiancePRD* getPRD() {
    const unsigned int u0 = optixGetPayload_0();
    const unsigned int u1 = optixGetPayload_1();
    return reinterpret_cast<RadiancePRD*>( unpackPointer( u0, u1 ) );
}

static __forceinline__ __device__ void traceRadiance(
        OptixTraversableHandle handle,
        float3                 ray_origin,
        float3                 ray_direction,
        float                  tmin,
        float                  tmax,
        RadiancePRD*           prd
        ) {
    // TODO: deduce stride from num ray-types passed in params
    unsigned int u0, u1;
    packPointer( prd, u0, u1 );
    optixTrace(
            handle,
            ray_origin,
            ray_direction,
            tmin,
            tmax,
            0.0f,                // rayTime
            OptixVisibilityMask( 1 ),
            OPTIX_RAY_FLAG_NONE,
            RAY_TYPE_RADIANCE,        // SBT offset
            RAY_TYPE_COUNT,           // SBT stride
            RAY_TYPE_RADIANCE,        // missSBTIndex
            u0, u1 );
}

static __forceinline__ __device__ void setPayloadOcclusion(bool occluded) {
    optixSetPayload_0(static_cast<unsigned int>(occluded));
}

static __forceinline__ __device__ bool traceOcclusion(
    OptixTraversableHandle handle,
    float3                 ray_origin,
    float3                 ray_direction,
    float                  tmin,
    float                  tmax,
    float3&                emission
)
{
    unsigned int occluded = 0u;
    unsigned int u0, u1, u2;
    optixTrace(
        handle,
        ray_origin,
        ray_direction,
        tmin,
        tmax,
        0.0f,                    // rayTime
        OptixVisibilityMask(1),
        OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT,
        RAY_TYPE_OCCLUSION,      // SBT offset
        RAY_TYPE_COUNT,          // SBT stride
        RAY_TYPE_OCCLUSION,      // missSBTIndex
        occluded,
        u0,
        u1,
        u2
        );
    emission = make_float3(uint_as_float(u0), uint_as_float(u1), uint_as_float(u2));
    return occluded;
}

extern "C" __global__ void __closesthit__occlusion() {
    HitGroupData* rt_data = (HitGroupData*)optixGetSbtDataPointer();
    setPayloadOcclusion(true);
    optixSetPayload_1(float_as_int(rt_data->emission_color.x));
    optixSetPayload_2(float_as_int(rt_data->emission_color.y));
    optixSetPayload_3(float_as_int(rt_data->emission_color.z));
}


extern "C" __global__ void __raygen__rg() {
    const int    w = params.width;
    const int    h = params.height;
    const float3 eye = params.eye;
    const float3 U = params.U;
    const float3 V = params.V;
    const float3 W = params.W;
    const uint3  idx = optixGetLaunchIndex();
    const int    subframe_index = params.subframe_index;
    float imageAspectRatio = 1.0f;
    SamplerState sampler(pcgHash(tea<4>(idx.y * w + idx.x, subframe_index)));
    float3 result = make_float3(0.0f);
    int i = params.samples_per_launch;
    do
    {
        // The center of each pixel is at fraction (0.5,0.5)
        const float2 subpixel_jitter = make_float2(randUniform(sampler), randUniform(sampler));

        const float2 fragcord = make_float2(static_cast<float>(idx.x) + subpixel_jitter.x, static_cast<float>(idx.y) + subpixel_jitter.y);
        const float3 rd = rayDir(make_float2(w, h), fragcord, params.fov, imageAspectRatio);
        float3 ray_direction = normalize(rd.x * U + rd.y * V + rd.z * W);
        float3 ray_origin = eye;

        RadiancePRD prd;
        prd.weight = make_float3(1.f);
        prd.countEmitted = true;
        prd.wasDelta = false;
        prd.done = false;
        prd.sampler = sampler;
        prd.direction = ray_direction;

        int depth = 0;
        for (;; )
        {
            prd.emitted = make_float3(0.f);
            prd.radiance = make_float3(0.f);
            traceRadiance(
                params.handle,
                ray_origin,
                ray_direction,
                0.000f,  // tmin       // TODO: smarter offset
                1e16f,  // tmax
                &prd);

            result += prd.emitted;
            result += prd.radiance;

            if (depth >= 17) {
                //result += make_float3(0.0, 10.0f, 0.0);
                break;
            }

            if (prd.done || depth >= 17) // TODO RR, variable for depth
                break;

            ray_origin = prd.origin;
            ray_direction = prd.direction;

            ++depth;
        }
    }     while (--i);

    const uint3    launch_index = optixGetLaunchIndex();
    const unsigned int image_index = launch_index.y * params.width + launch_index.x;
    float3         accum_color = result / static_cast<float>(params.samples_per_launch);

    if (subframe_index > 0)
    {
        const float                 a = 1.0f / static_cast<float>(subframe_index + 1);
        const float3 accum_color_prev = make_float3(params.accum_buffer[image_index]);
        accum_color = lerp(accum_color_prev, accum_color, a);
    }
    if (!isfinite(accum_color)) {
        params.accum_buffer[image_index] = make_float4(accum_color, 1.0f);
        params.frame_buffer[image_index] = make_color(make_float3(0.f, 0.0, 10.0));
        printf("asdfsaf\n");
    }
    else {
        params.accum_buffer[image_index] = make_float4(accum_color, 1.0f);
        params.frame_buffer[image_index] = make_color(accum_color);
    }
}

extern "C" __global__ void __miss__radiance() {
    MissData* rt_data  = reinterpret_cast<MissData*>( optixGetSbtDataPointer() );
    RadiancePRD* prd = getPRD();

    prd->radiance = make_float3(0.0, 0.0, 0.0);
    prd->emitted = make_float3(0.0f);
    prd->done      = true;
}

extern "C" __global__ void __closesthit__radiance() {
    HitGroupData* rt_data = (HitGroupData*)optixGetSbtDataPointer();

    const int    prim_idx        = optixGetPrimitiveIndex();
    const float3 ray_dir         = optixGetWorldRayDirection();
    const int    vert_idx_offset = prim_idx*3;

    const float3 v0   = make_float3( rt_data->vertices[ vert_idx_offset+0 ] );
    const float3 v1   = make_float3( rt_data->vertices[ vert_idx_offset+1 ] );
    const float3 v2   = make_float3( rt_data->vertices[ vert_idx_offset+2 ] );
    const float3 n0   = make_float3( rt_data->normals[ vert_idx_offset+0 ] );
    const float3 n1   = make_float3( rt_data->normals[ vert_idx_offset+1 ] );
    const float3 n2   = make_float3( rt_data->normals[ vert_idx_offset+2 ] );
    float2 bary = optixGetTriangleBarycentrics();
    float3 SN = normalize(bary.x * n1 + bary.y * n2 + (1.0f - bary.x - bary.y) * n0);
    const float3 N_0  = normalize( cross( v1-v0, v2-v0 ) );
    RadiancePRD* prd = getPRD();

    float3 N = N_0;
    if (dot(N, -ray_dir) < 0) {
        if (!isTransimissionBSDF(rt_data->bsdf.type())){
            prd->done = true;
            return;
        }
        if (rt_data->twofaced) {
            N *= -1.0f;
            SN *= -1.0f;
        }
    }
    const float3 P    = optixGetWorldRayOrigin() + optixGetRayTmax()*ray_dir;


    SamplerState sampler = prd->sampler;

    Onb onb(SN);
    Onb geoOnb(N);
    float3 wo = normalize(onb.transform(-ray_dir));


    float3 lightPos;
    float lightPdf;
    float3 lightEmission;
    float3 lightNormal;
    sampleLight(params.lightData, sampler, lightPos, lightPdf, lightEmission, lightNormal);


    float3 L = normalize(lightPos - P);
    float3 wL = onb.transform(L);
    float Ldist = length(lightPos - P);
    const float NoL = fabs(dot(SN, L));
    lightPdf *= Ldist * Ldist / fabs(dot(-L, lightNormal));
    float3 direct = make_float3(0.0f);
    {
        BSDFOutput lightBsdfRes;
        evalBSDF(params.bsdfData, rt_data->bsdf, wo, wL, lightBsdfRes);

        BSDFOutput bsdfRes;
        float3 wi;
        sampleBSDF(params.bsdfData, sampler, rt_data->bsdf, wo, wi, bsdfRes);
        float NoW = fabs(dot(wi, make_float3(0.0f, 0.0f, 1.0f)));
        onb.inverse_transform(wi);
        if ((dot(N, L) > 0.0f && dot(lightNormal, -L) > 0) || isTransimissionBSDF(rt_data->bsdf.type())) {
            float3 shadowEmission;
            bool occluded = traceOcclusion(
                params.handle,
                P,
                L,
                0.00001f,         // tmin
                Ldist - 0.01f,  // tmax
                shadowEmission
            );
            if (!occluded && isvalid(lightPdf) && isvalid(lightBsdfRes.bsdf) && lightPdf != 0.0f) {
                float w = powerHeuristic(1, lightPdf, 1, bsdfRes.pdf);
                direct +=  NoL * lightBsdfRes.bsdf * prd->weight * lightEmission / lightPdf;
            }
        }

        if (dot(wi, N) > 0.0f || isTransimissionBSDF(rt_data->bsdf.type())) {
            float3 shadowEmission;
            bool occluded = traceOcclusion(
                params.handle,
                P,
                wi,
                0.00001f,         // tmin
                1e16f,  // tmax
                shadowEmission
            );
        
            if (occluded && isvalid(bsdfRes.pdf) && isvalid(bsdfRes.bsdf) && bsdfRes.pdf != 0.0f) {
                float w = powerHeuristic(1, bsdfRes.pdf, 1, lightPdf);
                direct += bsdfRes.bsdf * shadowEmission * NoW * prd->weight * w / bsdfRes.pdf;
                if (rt_data->bsdf.type() == BSDF_ROUGH_PLASTIC) {
                    //printf("%d %f %f %f %f\n", rt_data->bsdf.type(), direct.x, direct.y, direct.z, bsdfRes.bsdf.x);
                }
            }
        }
    }
    prd->radiance = direct;

    BSDFOutput bsdfRes;
    float3 wi;
    sampleBSDF(params.bsdfData, sampler, rt_data->bsdf, wo, wi, bsdfRes);
    float NoW = abs(dot(wi, make_float3(0.0f, 0.0f, 1.0f)));
    onb.inverse_transform(wi);

    if( prd->countEmitted)
        prd->emitted = prd->weight * rt_data->emission_color;

    if (prd->wasDelta)
        prd->emitted = prd->weight * rt_data->emission_color;

    if (dot(wi, N) <= 0.0f && !isTransimissionBSDF(rt_data->bsdf.type())) {
        prd->done = true;
        return;
    }

    if (!isvalid(bsdfRes.pdf) || !isvalid(bsdfRes.bsdf) || bsdfRes.pdf == 0.0f) {
        prd->done = true;
        return;
    }

    prd->countEmitted = false;
    prd->origin = P + 0.0001f*faceforward(N, wi, N);
    prd->direction = wi;
    prd->weight *= bsdfRes.bsdf * NoW / bsdfRes.pdf;
    prd->wasDelta = bsdfRes.isDelta;
    prd->sampler = sampler;

}
