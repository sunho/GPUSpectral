#include "hip/hip_runtime.h"
#include <optix.h>
#include <hip/hip_vector_types.h>
#include <vector_functions.hpp>
#include "PathTracer.cuh"
#include "VectorMath.cuh"
#include <math.h>

extern "C" {
    __constant__ Params params;
}

struct RadiancePRD {
    float3       emitted;
    float3       radiance;
    float3       weight;
    float3       origin;
    float3       direction;
    float        directWeight;
    
    SamplerState sampler;
    int          wasDelta;
    int          countEmitted;
    int          done;
    int          pad;
};

static __forceinline__ __device__ void* unpackPointer( unsigned int i0, unsigned int i1 )
{
    const unsigned long long uptr = static_cast<unsigned long long>( i0 ) << 32 | i1;
    void*           ptr = reinterpret_cast<void*>( uptr );
    return ptr;
}

static __forceinline__ __device__ void  packPointer( void* ptr, unsigned int& i0, unsigned int& i1 )
{
    const unsigned long long uptr = reinterpret_cast<unsigned long long>( ptr );
    i0 = uptr >> 32;
    i1 = uptr & 0x00000000ffffffff;
}

static __forceinline__ __device__ RadiancePRD* getPRD() {
    const unsigned int u0 = optixGetPayload_0();
    const unsigned int u1 = optixGetPayload_1();
    return reinterpret_cast<RadiancePRD*>( unpackPointer( u0, u1 ) );
}

static __forceinline__ __device__ void traceRadiance(
        OptixTraversableHandle handle,
        float3                 ray_origin,
        float3                 ray_direction,
        float                  tmin,
        float                  tmax,
        RadiancePRD*           prd
        ) {
    // TODO: deduce stride from num ray-types passed in params
    unsigned int u0, u1;
    packPointer( prd, u0, u1 );
    optixTrace(
            handle,
            ray_origin,
            ray_direction,
            tmin,
            tmax,
            0.0f,                // rayTime
            OptixVisibilityMask( 1 ),
            OPTIX_RAY_FLAG_NONE,
            RAY_TYPE_RADIANCE,        // SBT offset
            RAY_TYPE_COUNT,           // SBT stride
            RAY_TYPE_RADIANCE,        // missSBTIndex
            u0, u1 );
}

static __forceinline__ __device__ void setPayloadOcclusion(bool occluded) {
    optixSetPayload_0(static_cast<unsigned int>(occluded));
}

static __forceinline__ __device__ bool traceOcclusion(
    OptixTraversableHandle handle,
    float3                 ray_origin,
    float3                 ray_direction,
    float                  tmin,
    float                  tmax
)
{
    unsigned int occluded = 0u;
    optixTrace(
        handle,
        ray_origin,
        ray_direction,
        tmin,
        tmax,
        0.0f,                    // rayTime
        OptixVisibilityMask(1),
        OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT,
        RAY_TYPE_OCCLUSION,      // SBT offset
        RAY_TYPE_COUNT,          // SBT stride
        RAY_TYPE_OCCLUSION,      // missSBTIndex
        occluded
        );
    return occluded;
}

extern "C" __global__ void __closesthit__occlusion() {
    setPayloadOcclusion(true);
}

extern "C" __global__ void __raygen__rg() {
    const int    w = params.width;
    const int    h = params.height;
    const float3 eye = params.camera.eye;
    const float3 U = params.camera.U;
    const float3 V = params.camera.V;
    const float3 W = params.camera.W;
    const uint3  idx = optixGetLaunchIndex();
    const int    subframe_index = params.subframeIndex;
    float imageAspectRatio = 1.0f;
    SamplerState sampler(pcgHash(tea<4>(idx.y * w + idx.x, params.subframeIndex)));
    float3 result = make_float3(0.0f);
    int i = params.spp;
    do {
        const float2 subpixel_jitter = make_float2(randUniform(sampler), randUniform(sampler));

        const float2 fragcord = make_float2(static_cast<float>(idx.x) + subpixel_jitter.x, static_cast<float>(idx.y) + subpixel_jitter.y);
        const float3 rd = rayDir(make_float2(w, h), fragcord, params.camera.fov, imageAspectRatio);
        float3 ray_direction = normalize(rd.x * U + rd.y * V + rd.z * W);
        float3 ray_origin = eye;

        RadiancePRD prd;
        prd.weight = make_float3(1.f);
        prd.countEmitted = true;
        prd.wasDelta = false;
        prd.done = false;
        prd.sampler = sampler;
        prd.direction = ray_direction;

        int depth = 0;
        for (;; )
        {
            prd.emitted = make_float3(0.f);
            prd.radiance = make_float3(0.f);
            traceRadiance(
                params.scene.tlas,
                ray_origin,
                ray_direction,
                0.000f,
                1e16f,
                &prd);

            result += prd.emitted;
            result += prd.radiance;

            if (depth >= 17) {
                break;
            }

            if (prd.done || depth >= 17) // TODO RR, variable for depth
                break;

            ray_origin = prd.origin;
            ray_direction = prd.direction;

            ++depth;
        }
    } while (--i);

    const uint3    launch_index = optixGetLaunchIndex();
    const unsigned int image_index = launch_index.y * params.width + launch_index.x;
    float3         accum_color = result / static_cast<float>(params.spp);

    if (subframe_index > 0)
    {
        const float                 a = 1.0f / static_cast<float>(subframe_index + 1);
        const float3 accum_color_prev = make_float3(params.accumBuffer[image_index]);
        accum_color = lerp(accum_color_prev, accum_color, a);
    }
    if (!isfinite(accum_color)) {
        params.accumBuffer[image_index] = make_float4(accum_color, 1.0f);
        params.frameBuffer[image_index] = make_color(make_float3(0.f, 0.0, 10.0));
        printf("nan detected in framebuffer\n");
    }
    else {
        params.accumBuffer[image_index] = make_float4(accum_color, 1.0f);
        params.frameBuffer[image_index] = make_color(accum_color);
    }
}

extern "C" __global__ void __miss__radiance() {
    MissData* rt_data  = reinterpret_cast<MissData*>( optixGetSbtDataPointer() );
    RadiancePRD* prd = getPRD();

    prd->radiance = make_float3(0.0, 0.0, 0.0);
    prd->emitted = make_float3(0.0f);
    prd->done      = true;
}

extern "C" __global__ void __closesthit__radiance() {
    HitGroupData* rt_data = (HitGroupData*)optixGetSbtDataPointer();

    const int    prim_idx        = optixGetPrimitiveIndex();
    const float3 ray_dir         = optixGetWorldRayDirection();
    const int    vert_idx_offset = prim_idx*3;

    const float3 v0   = make_float3( rt_data->vertices[ vert_idx_offset+0 ] );
    const float3 v1   = make_float3( rt_data->vertices[ vert_idx_offset+1 ] );
    const float3 v2   = make_float3( rt_data->vertices[ vert_idx_offset+2 ] );
    const float3 n0   = make_float3( rt_data->normals[ vert_idx_offset+0 ] );
    const float3 n1   = make_float3( rt_data->normals[ vert_idx_offset+1 ] );
    const float3 n2   = make_float3( rt_data->normals[ vert_idx_offset+2 ] );
    float2 bary = optixGetTriangleBarycentrics();
    float3 SN = normalize(bary.x * n1 + bary.y * n2 + (1.0f - bary.x - bary.y) * n0);
    const float3 N_0  = normalize( cross( v1-v0, v2-v0 ) );
    RadiancePRD* prd = getPRD();

    float3 N = N_0;
    if (dot(N, -ray_dir) < 0) {
        /*if (!isTransimissionBSDF(rt_data->bsdf.type())) {
            prd->done = true;
            return;
        }*/
        if (rt_data->twofaced) {
            N *= -1.0f;
            SN *= -1.0f;
        }
    }
    const float3 P    = optixGetWorldRayOrigin() + optixGetRayTmax()*ray_dir;
    SamplerState sampler = prd->sampler;

    Onb onb(SN);
    Onb geoOnb(N);
    float3 wo = normalize(onb.transform(-ray_dir));

    LightOutput lightRes;
    sampleLight(params.scene.lightData, sampler, &lightRes);

    float3 L = normalize(lightRes.position - P);
    float3 wL = onb.transform(L);
    float Ldist = length(lightRes.position - P);
    const float NoL = fabs(dot(SN, L));
    float lightPdf = Ldist * Ldist / fabs(dot(-L, lightRes.normal)) * lightRes.pdf;

    BSDFOutput bsdfRes;
    float3 wi;
    sampleBSDF(params.scene.bsdfData, sampler, rt_data->bsdf, wo, wi, bsdfRes);
    float NoW = abs(dot(wi, make_float3(0.0f, 0.0f, 1.0f)));
    onb.inverse_transform(wi);

    if (dot(wi, N) <= 0.0f && !isTransimissionBSDF(rt_data->bsdf.type())) {
        prd->done = true;
        return;
    }

    if (!isvalid(bsdfRes.pdf) || !isvalid(bsdfRes.bsdf) || bsdfRes.pdf == 0.0f) {
        prd->done = true;
        return;
    }

    float3 direct = make_float3(0.0f);
    if (!bsdfRes.isDelta) {
        BSDFOutput lightBsdfRes;
        evalBSDF(params.scene.bsdfData, rt_data->bsdf, wo, wL, lightBsdfRes);

        if ((dot(N, L) > 0.0f && dot(lightRes.normal, -L) > 0) || isTransimissionBSDF(rt_data->bsdf.type())) {
            bool occluded = traceOcclusion(
                params.scene.tlas,
                P,
                L,
                0.00001f,
                Ldist - 0.01f
            );
            if (!occluded && isvalid(lightPdf) && isvalid(lightBsdfRes.bsdf) && lightPdf != 0.0f) {
                float w = powerHeuristic(1, lightPdf, 1, bsdfRes.pdf);
                direct += w * NoL * lightBsdfRes.bsdf * prd->weight * lightRes.emission / lightPdf;
            }
        }
    }
    float lightFlag = dot(N, -ray_dir) > 0 ? 1.0f : 0.0f;
    if (prd->countEmitted) {
        prd->emitted = prd->weight * rt_data->emission_color * lightFlag;
    } else {
        if (prd->wasDelta) {
            prd->emitted = prd->weight * rt_data->emission_color * lightFlag;
        } else {
            direct += rt_data->emission_color * lightFlag * prd->directWeight * prd->weight;
        }
    }
    prd->radiance = direct;
    prd->directWeight = powerHeuristic(1, bsdfRes.pdf, 1, lightPdf);
    prd->countEmitted = false;
    prd->origin = P + 0.0001f*faceforward(N, wi, N);
    prd->direction = wi;
    prd->weight *= bsdfRes.bsdf * NoW / bsdfRes.pdf;
    prd->wasDelta = bsdfRes.isDelta;
    prd->sampler = sampler;
}
